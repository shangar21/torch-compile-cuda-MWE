#include "hip/hip_runtime.h"
#include <torch/extension.h>

__global__ void matmul_kernel(float* a, float* b, float* c, int D) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < D && col < D) {
        for (int k = 0; k < D; ++k) {
            c[row * D + col] += a[row * D + k] * b[k * D + col];
        }
    }
}

torch::Tensor matDxD(torch::Tensor a, torch::Tensor b) {
    const auto D = a.size(0);
    auto c = torch::zeros({D, D}, a.options());

    const int threads = 16;
    const dim3 blocks((D + threads - 1) / threads, (D + threads - 1) / threads);
    const dim3 threadsPerBlock(threads, threads);

    matmul_kernel<<<blocks, threadsPerBlock>>>(
        a.data_ptr<float>(),
        b.data_ptr<float>(),
        c.data_ptr<float>(),
        D
    );

    return c;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("matDxD", &matDxD, "Matrix multiplication on GPU");
}

